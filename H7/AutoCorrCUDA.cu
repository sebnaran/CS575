#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#ifndef BLOCKSIZE
#define BLOCKSIZE		128     // number of threads per block
#endif

#define NUMBLOCKS		256

__global__  void AutoCorr( float *dA, float *dSums )
{
	int gid   = blockIdx.x*blockDim.x + threadIdx.x;
	int shift = gid;

	float sum = 0.;
	for( int i = 0; i < 32768; i++ )
	{
		sum += dA[i] * dA[i + shift];
	}
	dSums[shift] = sum;
}

int
main( int argc, char *argv[ ] )
{

// Reading Data from file

FILE *fp = fopen( "signal.txt", "r" );
if( fp == NULL )
{
	fprintf( stderr, "Cannot open file 'signal.txt'\n" );
	exit( 1 );
}

int Size;
fscanf( fp, "%d", &Size );
float *hA =     new float[ 2*Size ];
float *hSums  = new float[ 1*Size ];
for( int i = 0; i < Size; i++ )
{
	fscanf( fp, "%f", &hA[i] );
	hA[i+Size] = hA[i];		// duplicate the array
}
fclose( fp );

// Allocate Device memory
hipError_t status;
float *dA, *dSums;

status = hipMalloc( (void **)(&dA), 2*Size*sizeof(float) );
checkCudaErrors( status );

status = hipMalloc( (void **)(&dSums), Size*sizeof(float) );
checkCudaErrors( status );

// copy host memory to device
status = hipMemcpy( dA, hA, 2*Size*sizeof(float), hipMemcpyHostToDevice );
checkCudaErrors( status );

status = hipMemcpy( dSums, hSums, Size*sizeof(float), hipMemcpyHostToDevice );
checkCudaErrors( status );

// setup the execution parameters:

dim3 threads(BLOCKSIZE, 1, 1 );
dim3 grid(   NUMBLOCKS, 1, 1 );

	// create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
	checkCudaErrors( status );
	status = hipEventCreate( &stop );
	checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
	checkCudaErrors( status );

	// execute the kernel:

	AutoCorr<<< grid,threads >>>(dA,dSums);
// record the stop event:

status = hipEventRecord( stop, NULL );
checkCudaErrors( status );

// wait for the stop event to complete:

status = hipEventSynchronize( stop );
checkCudaErrors( status );

float msecTotal = 0.0f;
status = hipEventElapsedTime( &msecTotal, start, stop );
checkCudaErrors( status );
double secondsTotal = 0.001 * (double)msecTotal;
//compute performance

double Performance = 2*(double)Size*(double)Size/ ( secondsTotal ) / 1000000.;
fprintf( stderr,"Performance = %8.2lf MegaOperations/Sec\n", Performance );

//Saving the Sums array in a matlab m file
// std::ofstream FileTemp;
// FileTemp.open("FSums.m");
// FileTemp << "Sums = [";
// FileTemp << Sums[0];
// for( int k = 1; k < Size; k++)
// {
// 	FileTemp << "," << Sums[k];
// }
// FileTemp<<"];";
// FileTemp.close();
}